
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Questo è il programma più semplice che si può immaginare scritto con CUDA C
 *
 * Cosa fa? Di pratico nulla. Chiama però il kernel "dark", dimostrando che qualcosa
 * di oscuro è accaduto sulla scheda grafica. Molte domande restano ancora in sospeso,
 * ma il modello di programmazione host-kernel dovrebbe a questo punto essere chiaro.
 *
 */

// Attenzione a questa parola chiave. Definisce un kernel, ovvero un processo che avviene
// sulla GPU
__global__ void dark(void)
{
	// Questo è un kernel vuoto. Può comunque essere invocato dal main.
}

int main(void)
{
	// Lancio il kernel "dark"
	dark<<<1,1>>>();
	printf("Benvenuto nel magico mondo delle GPU. Guarda il file 'hello_kernel.cu' per capire di cosa si tratta.\n");
	return 0;
}

