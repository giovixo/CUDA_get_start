/*
 * sarray.cu
 *
 *  Created on: 18/apr/2014
 *      Author: Giovanni De Cesare
 *
 *      Scopo: somma due vettori
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

// Attenzione a questa parola chiave. Definisce un kernel, ovvero un processo che avviene
// sulla GPU
__global__ void dark(void)
{
	// Oggi non mi va di fare nulla. E in effetti non faccio niente.
}

// Questo kernel calcola la somma di due vettori "a[N]" e "b[N]"
// I cores della GPU che vanno da 0 a N fanno una somma.
__global__ void add(int *a, int *b, int *c)
{
	int id = blockIdx.x; 
	if (id < N)
	{
		c[id]= a[id] + b[id];
	}
}

int main(void)
{
	int a[N], b[N], c[N]; // tre vettori a, b, c allocati sulla CPU
	int *dev_a, *dev_b, *dev_c; // tre vettori dev_[a,b,c] da allocare sulla GPU

	// Devo allocare la memoria della GPU per i tre vettori a, b, c
	hipMalloc( (void**)&dev_a, N * sizeof(int) );
	hipMalloc( (void**)&dev_b, N * sizeof(int) );
	hipMalloc( (void**)&dev_c, N * sizeof(int) );

	// Assegno ai vettori a, b dei valori (arbitrari)
	for (int i=0; i < N; i++)
	{
		a[i] = i;
		b[i] = i;
	}
	// Quindi copio a, b nella GPU
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );

	// lancio il kernel
	add<<<N,1>>>(dev_a, dev_b, dev_c);

	// L'array somma c sta ancora sulla GPU. Lo devo copiare sulla mamoria del processore
	// prima di poterlo usare
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );

	// Finalmente scrivo il risultato della somma
	for (int i=0; i<N; i++)
	{
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// E' sempre una buona abitudine liberare la memoria dopo averla usata
	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	printf("Bye.\n");

	return 0;
}

