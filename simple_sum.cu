
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Scopo: somma due interi
 *
 * Tasks:
 *  * Uso di un kernel
 *  * allocazione delle memoria GPU
 *  * Trasferimento di un intero dalla GPU al processore
 */

// Attenzione a questa parola chiave. Definisce un kernel, ovvero un processo che avviene
// sulla GPU
__global__ void dark(void)
{
	// Oggi non mi va di fare nulla. E in effetti non faccio niente.
}

// Come sopra, ma questa volta il kernel è estremamente complesso ;)
__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}

int main(void)
{
	int c;
	int *dev_c;

	// Devo allocare la memoria
	hipMalloc( (void**)&dev_c, sizeof(int) );

	// lancio il kernel
	add<<<1,1>>>(2,4, dev_c);

	// Il numerello sta ancora sulla GPU. Me lo devo copiare sulla mamoria del processore
	// prima di poterlo usare
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost );

	//printf("Benvenuto nel magico mondo delle GPU. Guarda il file 'somma.cu' per capire di cosa si tratta.\n");
	printf("2 + 4 = %d\n", c);

	// E' sempre una buona abitudine liberare la memoria dopo averla usata
	hipFree( dev_c );

	return 0;
}
